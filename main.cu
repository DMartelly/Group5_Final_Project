#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

int* generateAdjMatrix(int count, int* adjMatrix);
void printAdjMatrix(int count, int* adjMatrix);
int* multiplyMatrix(int* in,int* in2, int num,int count);
void matrixMultiplication(int count, int path, int* matrix);

#define NUMTHREADS 1024;

//This is the main function
int main(int argc, char* argv[]){
	int* adjMatrix = NULL;
//	int* gpuMatrix;
//	int* multipliedMatrix = NULL;
	int count;
	int path;
	if(argc > 3){
		 fprintf(stderr,"Usage: %s <node count>\n",argv[0]);
		 return 1;
	}
	if(argc==1){
	 	count = 10;
	 	path = 2;
	}
	else if(argc == 2){
		count = atoi(argv[1]);
		path = 2;
	}
	else{
	 	count = atoi(argv[1]);
		path = atoi(argv[2]);
	}
	adjMatrix = generateAdjMatrix(count, adjMatrix);
	matrixMultiplication(count, path, adjMatrix);	
	return 0;
}

__global__ void multiply(int* matrix, int* multipliedMatrix, int count){
        int element = blockIdx.x*blockDim.x + threadIdx.x;
	int sum = 0;
	int i;
	int col = element % count;
	int row = element / count;
	for(i=0; i < count; i++){
		sum+=matrix[count*i + col]*matrix[row*count + i];
	}
	multipliedMatrix[element] = sum;
}


//Prep for calling the gpu matrix multiplication function

void matrixMultiplication(int count, int path, int* matrix){
	int numThreads = NUMTHREADS;
	int* gpuMatrix;
	int* multipliedMatrix = (int*)malloc(count*count*sizeof(int));
	int* gpuMM;//gpu multiplied matrix
	int numBlocks;
	numBlocks = (count*count)/numThreads + 1;
        hipMalloc(&gpuMatrix, (count*count*sizeof(int)));
	hipMalloc(&gpuMM, (count*count*sizeof(int)));
        hipMemcpy(gpuMatrix, matrix, (count*count*sizeof(int)), hipMemcpyHostToDevice);

	multiply<<<numBlocks, numThreads>>>(gpuMatrix, gpuMM, count);

	hipMemcpy(matrix, gpuMatrix, (count*count*sizeof(int)), hipMemcpyDeviceToHost);
	hipMemcpy(multipliedMatrix, gpuMM, (count*count*sizeof(int)), hipMemcpyDeviceToHost);
        printAdjMatrix(count, matrix);
	printf("\n");
       // multipliedMatrix = multiplyMatrix(matrix,matrix,path,count);
        printAdjMatrix(count, multipliedMatrix);
	multipliedMatrix = multiplyMatrix(matrix,matrix,path,count);
	printf("\n");
	printAdjMatrix(count, multipliedMatrix);
}

//Creates an adjacency matrix
//	count - the size of the matrix. the size is count X count)
//	matrix - a pointer to an adjacency Matrix
int* generateAdjMatrix(int count, int* matrix){
	matrix = (int *)malloc(count*count*sizeof(int));
	int i, j;

	//Set the random seed to the current time
	srand(time(NULL));

	//Create a random adjacency matrix using rand
	for (i = 0; i < count; i++){
		for(j = 0; j < count; j++){
			if(i != j){
				int randomResult = rand() % 2;
				matrix[(i *count) + j] = randomResult;
				matrix[(j *count) + i] = randomResult;
			}
		}
	}
	return matrix;
}

//Returns a cross multiplied matrix of two matrixies
//	in - the first matrix
//	in2 - the second matrix
//	num - the number of times we do the multiplacation
//	size -
int* multiplyMatrix(int* in,int* in2,int num, int count){
	if(num==0)
		return in2;
	int arr[count];
	int i,j,k;
	int z,n=0;
	int* out = (int *) malloc(sizeof(int)*count*count);
	
	for(i=0; i<count; i++){
		for(j=0; j<count; j++){
			for(k=0;k<count;k++){
				arr[k] = in[(i*count)+k] * in2[(k*count)+j];
			}
			for(z=0;z<count;z++){
				n+=arr[z];	
			}
			out[(i*count)+j] = n;
			n=0;
		}
	}
	return multiplyMatrix(in,out,num-1,count);
}

//Prints the adjacency matrix to stdout
void printAdjMatrix(int count, int* matrix){
	int i;
	for (i = 0; i < count; i++){
		int j;
		for (j = 0; j < count; j++){
			printf("%i  ", matrix[(i * count) + j]);
		} 
		printf("\n");
	}
}

