#include <stdio.h>
#include <stdlib.h>

int* generateAdjMatrix(int count, int* adjMatrix);
void printAdjMatrix(int count, int* adjMatrix);
int* multiplyMatrix(int* in,int* in2, int num,int count);

//This is the main function
int main(int argc, char* argv[]){
	int* adjMatrix = NULL;
	int* multipliedMatrix = NULL;
	int count;
	int path;
	if(argc > 3){
		 fprintf(stderr,"Usage: %s <node count>\n",argv[0]);
		 return 1;
	}
	if(argc==1){
	 	count = 10;
	 	path = 2;
	}
	else if(argc == 2){
		count = atoi(argv[1]);
		path = 2;
	}
	else{
	 	count = atoi(argv[1]);
		path = atoi(argv[2]);
	}
	 
	adjMatrix = generateAdjMatrix(count, adjMatrix);
	
	hipMalloc(&adjMatrix, (count*count*sizeof(int)));
	
	printAdjMatrix(count, adjMatrix);
	multipliedMatrix = multiplyMatrix(adjMatrix,adjMatrix,path,count);
	printf("\n");
	printAdjMatrix(count, multipliedMatrix);
	return 0;
}

//Creates an adjacency matrix
//	count - the size of the matrix. the size is count X count)
//	matrix - a pointer to an adjacency Matrix
int* generateAdjMatrix(int count, int* matrix){
	matrix = (int *)malloc(count*count*sizeof(int));
	int i, j;

	//Set the random seed to the current time
	srand(time(NULL));

	//Create a random adjacency matrix using rand
	for (i = 0; i < count; i++){
		for(j = 0; j < count; j++){
			if(i != j){
				int randomResult = rand() % 2;
				matrix[(i *count) + j] = randomResult;
				matrix[(j *count) + i] = randomResult;
			}
		}
	}
	return matrix;
}

//Square a given matrix: in.
int* multiplyMatrix(int* in,int* in2,int num, int count){
	if(num==0) return in2;
	int arr[count];
	int i,j,k;
	int z,n=0;
	int* out = (int *) malloc(sizeof(int)*count*count);
	
	for(i=0; i<count; i++){
		for(j=0; j<count; j++){
			for(k=0;k<count;k++){
				arr[k] = in[(i*count)+k] * in2[(k*count)+j];
			}
			for(z=0;z<count;z++){
				n+=arr[z];	
			}
			out[(i*count)+j] = n;
			n=0;
		}
	}
	return multiplyMatrix(in,out,num-1,count);
}

//Prints the adjacency matrix to stdout
void printAdjMatrix(int count, int* matrix){
	int i;
	for (i = 0; i < count; i++){
		int j;
		for (j = 0; j < count; j++){
			printf("%i  ", matrix[(i * count) + j]);
		} 
		printf("\n");
	}
}

