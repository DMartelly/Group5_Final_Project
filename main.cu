#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

int* generateAdjMatrix(int count, int* adjMatrix);
void printAdjMatrix(int count, int* adjMatrix);
int* multiplyMatrix(int* in,int* in2, int num,int count);
void matrixMultiplication(int count, int path, int* matrix);

#define NUMTHREADS 1024;

//This is the main function
int main(int argc, char* argv[]){
	int* adjMatrix = NULL;
	//int* gpuMatrix;
	//int* multipliedMatrix = NULL;
	int count;
	int path;
	
	//If there is more than 2 parameters
	if(argc > 3){
		 fprintf(stderr,"Usage: %s <node count>\n",argv[0]);
		 return 1;
	}
	//If there are no parameters
	if(argc==1){
	 	count = 10;
	 	path = 2;
	}
	//If there is only one parameter
	else if(argc == 2){
		count = atoi(argv[1]);
		path = 2;
	}
	//If 2 parameters are given
	else{
	 	count = atoi(argv[1]);
		path = atoi(argv[2]);
	}

	//adjMatrix now equals a new Random adjancency  Matrix
	adjMatrix = generateAdjMatrix(count, adjMatrix);

	//Compute the GPU function
	matrixMultiplication(count, path, adjMatrix);	
	return 0;
}

__global__ void multiply(int* matrix, int* multipliedMatrix, int count){
        int element = blockIdx.x*blockDim.x + threadIdx.x;
	int sum = 0;
	int i;
	int col = element % count;
	int row = element / count;
	for(i=0; i < count; i++){
		sum+=matrix[count*i + col]*matrix[row*count + i];
	}
	multipliedMatrix[element] = sum;
}


//Prep for calling the gpu matrix multiplication function
void matrixMultiplication(int count, int path, int* matrix){
	
	int numThreads = NUMTHREADS;
	
	//An adjacency matrix on the GPU
	int* gpuMatrix;

	//The multiplied matrix on the GPU
	int* gpuMM;

	//A matrix that will store gpuMM on the CPU
	int* multipliedMatrix = (int*)malloc(count*count*sizeof(int));

	//The number of GPUS
	int numBlocks = (count*count)/numThreads + 1;

	//Allocate the memory on the GPU
        hipMalloc(&gpuMatrix, (count*count*sizeof(int)));
	hipMalloc(&gpuMM, (count*count*sizeof(int)));

	//Copy the input matrix from the CPU to the GPU (matrix -> gpuMatrix)
        hipMemcpy(gpuMatrix, matrix, (count*count*sizeof(int)), hipMemcpyHostToDevice);

	//Preform the multiplied matrix function on gpuMatrix and store into gpuMM
	multiply<<<numBlocks, numThreads>>>(gpuMatrix, gpuMM, count);

	//Copy gpuMM from the GPU to the CPU in multipiedMatrix
	hipMemcpy(multipliedMatrix, gpuMM, (count*count*sizeof(int)), hipMemcpyDeviceToHost);
        
	//Print the input matrix
	printAdjMatrix(count, matrix);
	printf("\n");

	//Print the multiplied matrix, copied earlier from the GPU
        printAdjMatrix(count, multipliedMatrix);
	print("\n");
	

	multipliedMatrix = multiplyMatrix(matrix,matrix,path,count);
	printf("\n");
	printAdjMatrix(count, multipliedMatrix);
}

//Creates an adjacency matrix
//	count - the size of the matrix. the size is count X count)
//	matrix - a pointer to an adjacency Matrix
int* generateAdjMatrix(int count, int* matrix){
	matrix = (int *)malloc(count*count*sizeof(int));
	int i, j;

	//Set the random seed to the current time
	srand(time(NULL));

	//Create a random adjacency matrix using rand
	for (i = 0; i < count; i++){
		for(j = 0; j < count; j++){
			if(i != j){
				int randomResult = rand() % 2;
				matrix[(i *count) + j] = randomResult;
				matrix[(j *count) + i] = randomResult;
			}
		}
	}
	return matrix;
}

//Returns a cross multiplied matrix of two matrixies
//	in - the first matrix
//	in2 - the second matrix
//	num - the number of times we do the multiplacation
//	size -
int* multiplyMatrix(int* in,int* in2,int num, int count){
	if(num==0)
		return in2;
	int arr[count];
	int i,j,k;
	int z,n=0;
	int* out = (int *) malloc(sizeof(int)*count*count);
	
	for(i=0; i<count; i++){
		for(j=0; j<count; j++){
			for(k=0;k<count;k++){
				arr[k] = in[(i*count)+k] * in2[(k*count)+j];
			}
			for(z=0;z<count;z++){
				n+=arr[z];	
			}
			out[(i*count)+j] = n;
			n=0;
		}
	}
	return multiplyMatrix(in,out,num-1,count);
}

//Prints the adjacency matrix to stdout
void printAdjMatrix(int count, int* matrix){
	int i;
	for (i = 0; i < count; i++){
		int j;
		for (j = 0; j < count; j++){
			printf("%i  ", matrix[(i * count) + j]);
		} 
		printf("\n");
	}
}

